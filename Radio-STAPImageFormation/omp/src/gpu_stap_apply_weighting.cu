#include "hip/hip_runtime.h"
/*
   This CUDA code is implementation of the kernel 
   space time adaptive processing - inner product
   as per C code in file stap_apply_weighting.c 
   provided with header as above

   The cuda implementation done by the team at PNNL
 */


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#include "stap_utils.h"

#define CUDA_SAFE(x) if ( hipSuccess != (x) ) { printf("CUDA CALL FAILED AT %d\n", __LINE__ ); exit(1);}
#define CUDA_SAFE_MALLOC(DP, SIZE)  (hipMalloc((void**)&DP, SIZE))

__constant__ complex steering_vectors[N_STEERING * (N_CHAN*TDOF)];

__global__ void gpu_compute_gamma_weights (complex *datacube, complex *adaptive_weights, complex *output)
{

		__shared__ complex shared_accum[N_CHAN*TDOF];
		__shared__ complex shared_accum_sv[N_STEERING];
		__shared__ float shared_gamma[N_STEERING];
		__shared__ complex shared_snapshot[N_CHAN*TDOF];

		//thread ID for given bock size and number of blocks
		int thread_number = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x * blockDim.y
				+ (threadIdx.y * blockDim.x) + threadIdx.x;

		if(thread_number >= N_DOP * N_BLOCKS * N_CHAN * TDOF) return;

		int block = blockIdx.x;
		if(block >= N_BLOCKS) return;

		int dop_index = blockIdx.y;
		if(dop_index >= N_DOP) return;

		if(threadIdx.x >= N_CHAN * TDOF) return;

		int _i = threadIdx.x;

		int offset;

		shared_gamma[_i] = 0.0f;
		shared_accum_sv[_i].re = 0.0f;
		shared_accum_sv[_i].im = 0.0f;
		if((N_CHAN*TDOF) < N_STEERING)
		{
				if(threadIdx.x == 0) 
						for(int m = (N_CHAN*TDOF); m < N_STEERING; m++) 
						{
								shared_gamma[m] = 0.0f;
								shared_accum_sv[m].re = 0.0f;
								shared_accum_sv[m].im = 0.0f;
						}
		}
		__syncthreads();


		int sv;
		for (sv = 0; sv < N_STEERING; ++sv)
		{
				//accum.re = accum.im = 0.0f;
				shared_accum[_i].re = 0.0f;
				shared_accum[_i].im = 0.0f;

				//const complex prod = cmult(cconj(adaptive_weights[dop_index][range_block][sv][i]),steering_vectors[sv][i]);
				shared_accum[_i].re = adaptive_weights[dop_index*(N_BLOCKS*N_STEERING*N_CHAN*TDOF)+block*(N_STEERING*N_CHAN*TDOF)+sv*(N_CHAN*TDOF)+_i].re 
						* steering_vectors[sv*(N_CHAN*TDOF)+_i].re 
						+ adaptive_weights[dop_index*(N_BLOCKS*N_STEERING*N_CHAN*TDOF)+block*(N_STEERING*N_CHAN*TDOF)+sv*(N_CHAN*TDOF)+_i].im 
						* steering_vectors[sv*(N_CHAN*TDOF)+_i].im;

				shared_accum[_i].im = adaptive_weights[dop_index*(N_BLOCKS*N_STEERING*N_CHAN*TDOF)+block*(N_STEERING*N_CHAN*TDOF)+sv*(N_CHAN*TDOF)+_i].re 
						* steering_vectors[sv*(N_CHAN*TDOF)+_i].im 
						- adaptive_weights[dop_index*(N_BLOCKS*N_STEERING*N_CHAN*TDOF)+block*(N_STEERING*N_CHAN*TDOF)+sv*(N_CHAN*TDOF)+_i].im 
						* steering_vectors[sv*(N_CHAN*TDOF)+_i].re;

				__syncthreads();

				if(threadIdx.x == 0) 
				{
						for(int j = 0; j < N_CHAN*TDOF; j++)
						{
								shared_accum_sv[sv].re += shared_accum[j].re;
								shared_accum_sv[sv].im += shared_accum[j].im;
						}
				}
				__syncthreads();

		}

		offset = N_STEERING / 2;

		if(threadIdx.x < offset)
		{
				for (int j = 0; j < 2; j++)
				{
						shared_gamma[threadIdx.x+j*offset] = sqrt(shared_accum_sv[threadIdx.x+j*offset].re * shared_accum_sv[threadIdx.x+j*offset].re 
										+ shared_accum_sv[threadIdx.x+j*offset].im * shared_accum_sv[threadIdx.x+j*offset].im);

						if (shared_gamma[threadIdx.x+j*offset] > 0)
						{
								shared_gamma[threadIdx.x+j*offset] = 1.0f / shared_gamma[threadIdx.x+j*offset];
						}
						else
						{
								shared_gamma[threadIdx.x+j*offset] = 1.0f;
						}
				}
		}
		__syncthreads();

		int first_cell = block*TRAINING_BLOCK_SIZE;
		int last_cell = (block+1)*TRAINING_BLOCK_SIZE-1;

		for (int cell = first_cell; cell <= last_cell; ++cell)
		{
				int dof;
				int chan = threadIdx.x;

				if(chan < N_CHAN)
				{
						for (dof = 0; dof < TDOF; ++dof)
						{
								int dop = dop_index - (TDOF-1)/2 + dof;
								if (dop < 0) { dop += N_DOP; }
								if (dop >= N_DOP) { dop -= N_DOP; }

								//snapshot[chan*TDOF+dof] = datacube[chan][dop][range_cell];
								shared_snapshot[chan*TDOF+dof] = datacube[chan*(N_DOP*N_RANGE)+dop*N_RANGE+cell];
						}
				}

				__syncthreads();

				int sv;
				for(sv = 0; sv < N_STEERING; ++sv)
				{
						shared_accum[_i].re = 0.0f;
						shared_accum[_i].im = 0.0f;

						shared_accum[_i].re = adaptive_weights[dop_index*(N_BLOCKS*N_STEERING*N_CHAN*TDOF)+block*(N_STEERING*N_CHAN*TDOF)+sv*(N_CHAN*TDOF)+_i].re * shared_snapshot[_i].re 
								+ adaptive_weights[dop_index*(N_BLOCKS*N_STEERING*N_CHAN*TDOF)+block*(N_STEERING*N_CHAN*TDOF)+sv*(N_CHAN*TDOF)+_i].im * shared_snapshot[_i].im;

						shared_accum[_i].im = adaptive_weights[dop_index*(N_BLOCKS*N_STEERING*N_CHAN*TDOF)+block*(N_STEERING*N_CHAN*TDOF)+sv*(N_CHAN*TDOF)+_i].re * shared_snapshot[_i].im
								- adaptive_weights[dop_index*(N_BLOCKS*N_STEERING*N_CHAN*TDOF)+block*(N_STEERING*N_CHAN*TDOF)+sv*(N_CHAN*TDOF)+_i].im * shared_snapshot[_i].re;

						__syncthreads();

						if(threadIdx.x == 0) 
						{
								for(int j = 1; j < N_CHAN*TDOF; j++)
								{
										shared_accum[0].re += shared_accum[j].re;
										shared_accum[0].im += shared_accum[j].im;
								}
						}
						__syncthreads();

						if(threadIdx.x == 0) output[sv*(N_DOP*N_RANGE)+dop_index*N_RANGE+cell].re = shared_accum[0].re * shared_gamma[sv];
						if(threadIdx.x == 0) output[sv*(N_DOP*N_RANGE)+dop_index*N_RANGE+cell].im = shared_accum[0].im * shared_gamma[sv];
				}
		}

}


extern "C" int gpu_stap_apply_weighting( 
    complex output[N_STEERING][N_DOP][N_RANGE],
    complex (* const datacube)[N_DOP][N_RANGE],
    complex (* const adaptive_weights)[N_BLOCKS][N_STEERING][N_CHAN*TDOF],
    complex (* const _steering_vectors)[N_CHAN*TDOF])
{
		complex *dev_datacube;
		complex *dev_adaptive_weights;
		complex *dev_output;

		const int num_datacube = N_CHAN * N_DOP * N_RANGE;
		const int num_adaptive_weight_elements = N_DOP * N_BLOCKS * N_STEERING * (N_CHAN*TDOF);
		const int num_steering_vector_elements = N_STEERING * (N_CHAN*TDOF);
		const int num_output_elements = N_STEERING * N_DOP * N_RANGE;


		CUDA_SAFE(hipMalloc((void **)&dev_datacube, num_datacube * sizeof(complex)));
		CUDA_SAFE(hipMalloc((void **)&dev_adaptive_weights, num_adaptive_weight_elements * sizeof(complex)));
		CUDA_SAFE(hipMalloc((void **)&dev_output, num_output_elements * sizeof(complex)));

		//copy data from host to device or initialize device variables
		CUDA_SAFE(hipMemcpy(dev_datacube, datacube, num_datacube * sizeof(complex), hipMemcpyHostToDevice));
		CUDA_SAFE(hipMemcpy(dev_adaptive_weights, adaptive_weights, num_adaptive_weight_elements * sizeof(complex), hipMemcpyHostToDevice));
		CUDA_SAFE(hipMemcpyToSymbol(HIP_SYMBOL(steering_vectors), _steering_vectors, num_steering_vector_elements * sizeof(complex), 0, hipMemcpyHostToDevice));
		CUDA_SAFE(hipMemset(dev_output, 0, num_output_elements * sizeof(complex)));

		//compute number of blocks for kernel launch
		dim3 grid_dim_01(N_BLOCKS,N_DOP,1);
		dim3 block_dim_01(N_CHAN * TDOF);

		gpu_compute_gamma_weights<<<grid_dim_01,block_dim_01>>>(dev_datacube, dev_adaptive_weights, dev_output);

		CUDA_SAFE(hipDeviceSynchronize());

		//copy variable out back from Device to Host
		CUDA_SAFE(hipMemcpy(output, dev_output, (num_output_elements * sizeof(complex)), hipMemcpyDeviceToHost));

		//free device variables
		CUDA_SAFE(hipFree(dev_datacube));
		CUDA_SAFE(hipFree(dev_adaptive_weights));
		CUDA_SAFE(hipFree(dev_output));

		return 0;
}
